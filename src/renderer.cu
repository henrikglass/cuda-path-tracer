#include "hip/hip_runtime.h"
#include "renderer.cuh"
#include "util.cuh"
#include "vector.cuh"
#include <stdio.h>


vec3 render(const vec3& a, const vec3& b) {
    int devID = 0;
    hipDeviceProp_t props;

    //Get GPU information
    hipGetDevice(&devID);
    hipGetDeviceProperties(&props, devID);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);

    vec3 ans;
    vec3 *d_ans;
    hipMalloc(&d_ans, sizeof(vec3));
    // call ladug
    ladug<<<1, 2>>>(d_ans, a, b);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(&ans, d_ans, sizeof(vec3), hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    return ans;
}

__global__
void ladug(vec3 *ans, const vec3 a, const vec3 b) {
    *ans = a + b;
    printf("hello\n");
}