#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "renderer.cuh"
#include "io.cuh"

int main(void) {
    // materials
    Material m_white(vec3(1,1,1), vec3(0.0f), 0, 0, false);
    Material m_lg(vec3(0.8f), vec3(0.0f), 0, 0, true);
    Material m_ball(vec3(0.8f,0.8f,0.8f), vec3(0.5f), 0, 0.0f, false);
    Material m_light(vec3(1,1,1), vec3(0.0f), 10, 0, false);
    Material m_default;
    Material m_hc;
    //m_hc.specular = vec3(0.2f);
    //m_hc.smoothness = 1.0f;
    //m_hc.albedo_map.set("examples/hcandersen/textures/diff.jpg");
    //m_hc.smoothness_map.set("examples/hcandersen/textures/gloss.jpg");
    m_hc.normal_map.set("examples/hcandersen/textures/normal.jpg");

    // entities
    Entity floor("examples/cornellbox/floor.obj", &m_default);
    Entity ball(vec3(10, 40, 30), 3.0f, &m_light);
    Entity hcandersen("examples/hcandersen/source/80k.obj", &m_hc);

    // transforms
    floor.scale(100.1f);
    hcandersen.rotate(vec3(-3.14159265f / 2.0f, 3.14159265f / 2.0f, 0));
    hcandersen.translate(vec3(0, 22.65, 0));

    // construct octree
    hcandersen.construct_octree();

    // debug
    hcandersen.print();

    // add to scene scene
    Scene scene;
    scene.add_entity(&hcandersen);
    //scene.add_entity(&ball);
    //scene.add_entity(&floor);

    //scene.set_hdri("examples/hdris/pink_sunrise_4k.hdr");
    scene.use_hdri_smoothing(true);
    scene.set_hdri("examples/hdris/quattro_canti_4k.hdr");
    scene.rotate_hdri(0.85f);

    // camera
    //ivec2 res = ivec2(1024, 768);
    ivec2 res = ivec2(1920, 1080);
    
    //Camera camera(vec3(40.5f, 20.5f, -50), vec3(-0.5, 0, 1).normalized(), res);
    //camera.focal_length = 1.5f;
    //camera.aperture = 1;
    //camera.focus_distance = 85;
    Camera camera(vec3(40.5f, 30.5f, -50), vec3(-0.5, 0, 1).normalized(), res);
    camera.focal_length = 2.5f;
    //camera.focal_length = 1.0f;
    camera.aperture = 1;
    camera.focus_distance = 85;

    // render
    Renderer renderer;
    //renderer.set_samples_per_pixel(1024);
    renderer.set_samples_per_pixel(2048);
    Image image = renderer.render(camera, scene);
    std::cout << image.resolution << std::endl;
    save_ppm("output.ppm", image);
}
