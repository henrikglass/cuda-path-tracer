#include "texture.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

//
// @Incomplete Doesn't adhere to the rule-of-three.
//

Texture::~Texture() {
    if (this->data != nullptr) {
        stbi_image_free(this->data);
    }
}

/**
 * Frees texture from device memory.
 */
void Texture::free_from_device() {
    if (this->d_data != nullptr) {
        gpuErrchk(hipFree(this->d_data));
        this->d_data = nullptr;
    }
}

/**
 * Copies color texture to device memory.
 */
void ColorTexture::copy_to_device() {
    if (!this->is_set())
        return;
    long size = sizeof(unsigned char) * resolution.x * resolution.y * 3;
    gpuErrchk(hipMalloc(&this->d_data, size));
    hipMemcpy(this->d_data, this->data, size, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
}

/**
 * Copies grayscale texture to device memory.
 */
void GrayscaleTexture::copy_to_device() {
    if (!this->is_set())
        return;
    
    long size = sizeof(unsigned char) * resolution.x * resolution.y;
    gpuErrchk(hipMalloc(&this->d_data, size));
    hipMemcpy(this->d_data, this->data, size, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
}

/**
 * Loads an image file and sets it as a color texture.
 * 
 * @param path      a path to a (*.jpg, *.png, *.gif, etc.) image-file.
 */
void ColorTexture::set(const std::string &path) {
    int n;
    this->data = stbi_load(path.c_str(), &(this->resolution.x), &(this->resolution.y), &n, 3);
}

/**
 * Loads an image file and sets it as a color texture.
 * 
 * @param path      a path to a (*.jpg, *.png, *.gif, etc.) image-file.
 */
void GrayscaleTexture::set(const std::string &path) {
    int n;
    this->data = stbi_load(path.c_str(), &(this->resolution.x), &(this->resolution.y), &n, 1);
}
