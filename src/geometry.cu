#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include <math.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include "util.cuh"

#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"

/************************************************************************************/
/*                                    Constructors                                  */
/************************************************************************************/

Entity::Entity(const std::string &path, const Material &material) {
    tinyobj::attrib_t attrib;
    std::vector<tinyobj::shape_t> shapes;
    std::vector<tinyobj::material_t> materials;
    std::string warn;
    std::string err;

    bool ret = tinyobj::LoadObj(&attrib, &shapes, &materials, &warn, &err,  path.c_str());

    if(!warn.empty()) {
        std::cout << warn << std::endl;
    }

    if(!err.empty()) {
        std::cout << err << std::endl;
    }

    if (!ret || shapes.size() != 1) {
        exit(1);
    }

    bool no_normals = (attrib.normals.size() / 3) == 0;

    // allocate space
    this->n_vertices  = attrib.vertices.size() / 3;
    this->n_triangles = shapes[0].mesh.num_face_vertices.size();
    this->vertices  = new Vertex[this->n_vertices];
    this->triangles = new Triangle[this->n_triangles];
    
    // For AABB
    vec3 min(FLT_MAX, FLT_MAX, FLT_MAX);
    vec3 max(-FLT_MAX, -FLT_MAX, -FLT_MAX);

    // load vertices
    size_t index_offset = 0;
    for (size_t v = 0; v < this->n_vertices; v++) {
        vec3 position(
                (float) attrib.vertices[index_offset + 0],
                (float) attrib.vertices[index_offset + 1],
                (float) attrib.vertices[index_offset + 2]
        );

        vec3 normal(0.0f, 0.0f, 0.0f);
        if(!no_normals) {
            normal.x = attrib.normals[index_offset + 0];
            normal.y = attrib.normals[index_offset + 1];
            normal.z = attrib.normals[index_offset + 2];
            normal.normalize();
        }

        // update min, max for aabb
        min.x = (position.x < min.x) ? position.x : min.x;
        min.y = (position.y < min.y) ? position.y : min.y;
        min.z = (position.z < min.z) ? position.z : min.z;
        max.x = (position.x > max.x) ? position.x : max.x;
        max.y = (position.y > max.y) ? position.y : max.y;
        max.z = (position.z > max.z) ? position.z : max.z;

        this->vertices[v] = Vertex(position, normal);
        index_offset += 3;
    }

    // load triangles
    tinyobj::shape_t shape = shapes[0];
    index_offset = 0;
    for (size_t f = 0; f < this->n_triangles; f++) {
        if ((int)shape.mesh.num_face_vertices[f] != 3) {
            std::cerr << "OBJ file faces must be triangles" << std::endl;
            exit(1);
        }

        this->triangles[f] = Triangle(
                shape.mesh.indices[index_offset + 0].vertex_index,
                shape.mesh.indices[index_offset + 1].vertex_index,
                shape.mesh.indices[index_offset + 2].vertex_index
        );

        if (no_normals) {
            int v0_idx = this->triangles[f].idx_a;
            int v1_idx = this->triangles[f].idx_b;
            int v2_idx = this->triangles[f].idx_c;
            vec3 v0 = this->vertices[v0_idx].position;
            vec3 v1 = this->vertices[v1_idx].position;
            vec3 v2 = this->vertices[v2_idx].position;
            vec3 e1 = v1 - v0;
            vec3 e2 = v2 - v0;
            vec3 t_normal = cross(e1, e2).normalized();
            this->vertices[v0_idx].normal = this->vertices[v0_idx].normal + t_normal;
            this->vertices[v1_idx].normal = this->vertices[v1_idx].normal + t_normal;
            this->vertices[v2_idx].normal = this->vertices[v2_idx].normal + t_normal;
        }

        index_offset += 3;
    }

    // normalize normals
    if (no_normals) {
        for (size_t i = 0; i < this->n_vertices; i++) {
            this->vertices[i].normal.normalize();
        }
    }

    // create AABB
    this->aabb = AABB(min, max);

    // set center
    this->center = vec3(
            (min.x + max.x) / 2,
            (min.y + max.y) / 2,
            (min.z + max.z) / 2
    );

    // set shape & material
    this->material = material;
    this->shape = TRIANGLE_MESH;
}

Entity::Entity(const vec3 &center, float radius, const Material &material) {
    this->shape     = SPHERE;
    this->center    = center;
    this->radius    = radius;
    this->material  = material;
}

/************************************************************************************/
/*                                 Misc. fucntions                                  */
/************************************************************************************/
void AABB::recalculate(Vertex *vertices, int n_vertices) {
    vec3 min(FLT_MAX, FLT_MAX, FLT_MAX);
    vec3 max(-FLT_MAX, -FLT_MAX, -FLT_MAX);
    for (int i = 0; i < n_vertices; i++) {
        vec3 position = vertices[i].position;
        min.x = (position.x < min.x) ? position.x : min.x;
        min.y = (position.y < min.y) ? position.y : min.y;
        min.z = (position.z < min.z) ? position.z : min.z;
        max.x = (position.x > max.x) ? position.x : max.x;
        max.y = (position.y > max.y) ? position.y : max.y;
        max.z = (position.z > max.z) ? position.z : max.z;
    }
    
    this->min = min;
    this->max = max;
}

void Entity::print() {
    std::cout << "center: " << this->center << std::endl;
    std::cout << "x: " << this->aabb.min.x << " to " << this->aabb.max.x << std::endl;
    std::cout << "y: " << this->aabb.min.y << " to " << this->aabb.max.y << std::endl;
    std::cout << "z: " << this->aabb.min.z << " to " << this->aabb.max.z << std::endl;
}

/************************************************************************************/
/*                            Geometric transformations                             */
/************************************************************************************/

void Entity::scale(float factor) {
    if (this->shape == SPHERE) {
        radius *= factor;
        return;
    }

    if (this->shape == TRIANGLE_MESH) {
        int v_size = this->n_vertices;
        for(int i = 0; i < v_size; i++) {
            vec3 pos = vertices[i].position;
            pos = ((pos - this->center) * factor) + this->center;
            vertices[i].position = pos;
        }

        // recalculate aabb
        this->aabb.recalculate(this->vertices, this->n_vertices);
    }
}

void Entity::translate(vec3 delta) {
    // move center for all shapes
    this->center = this->center + delta;

    if (this->shape == TRIANGLE_MESH) {
        int v_size = this->n_vertices;
        for(int i = 0; i < v_size; i++) {
            vec3 pos = vertices[i].position;
            pos = pos + delta;
            vertices[i].position = pos;
        }

        // recalculate aabb
        this->aabb.recalculate(this->vertices, this->n_vertices);
    }
}

void Entity::rotate(vec3 rot) {
    if (this->shape == SPHERE)
        return;

    // a = alpha, b = beta, g = gamma. For rotation on z, y and x respectively.
    float cos_a = cos(rot.z);
    float sin_a = sin(rot.z);
    float cos_b = cos(rot.y);
    float sin_b = sin(rot.y);
    float cos_g = cos(rot.x);
    float sin_g = sin(rot.x);

    // Rotation matrix R:
    vec3 R0(/*[0,0]*/ cos_a * cos_b, /*[1,0]*/  cos_a*sin_b*sin_g - sin_a*cos_g, /*[2,0]*/  cos_a*sin_b*cos_g + sin_a*sin_g);
    vec3 R1(/*[0,1]*/ sin_a * cos_b, /*[1,1]*/  sin_a*sin_b*sin_g + cos_a*cos_g, /*[2,1]*/  sin_a*sin_b*cos_g - cos_a*sin_g);
    vec3 R2(/*[0,2]*/ -sin_b,        /*[1,2]*/  cos_b*sin_g,                     /*[2,2]*/  cos_b*cos_g);
    
    // rotate on x
    for (size_t i = 0; i < this->n_vertices; i++) {
        // rotate vertex positions
        vec3 v = this->vertices[i].position - this->center;
        v = vec3(dot(v, R0), dot(v, R1), dot(v, R2));
        this->vertices[i].position = v + this->center;

        // rotate vertex normals
        vec3 n = this->vertices[i].normal;
        n = vec3(dot(n, R0), dot(n, R1), dot(n, R2));
        this->vertices[i].normal = n;
    }

    // recalculate aabb
    this->aabb.recalculate(this->vertices, this->n_vertices);

    // TODO rotate on y and z. Preferable not one at a time.
}

/************************************************************************************/
/*                                Memory management                                 */
/************************************************************************************/

void Entity::copy_to_device() {
    if (this->shape == SPHERE)
        return;

    if (this->shape == TRIANGLE_MESH) {
        // copy vertices
        long vertices_size = this->n_vertices * sizeof(Vertex);
        gpuErrchk(hipMalloc(&this->d_vertices, vertices_size));
        hipMemcpy(this->d_vertices, this->vertices, vertices_size, hipMemcpyHostToDevice);
        gpuErrchk(hipPeekAtLastError());

        // copy triangles
        long triangles_size = this->n_triangles * sizeof(Triangle);
        gpuErrchk(hipMalloc(&this->d_triangles, triangles_size));
        hipMemcpy(this->d_triangles, this->triangles, triangles_size, hipMemcpyHostToDevice);
        gpuErrchk(hipPeekAtLastError());
    }
}

void Entity::free_from_device() {
    if (this->shape == SPHERE)
        return;

    // TODO implement mesh case
}


/************************************************************************************/
/*                            Intersection functions                                */
/************************************************************************************/

__device__
bool get_closest_intersection_in_scene(const Ray &ray, Entity *entities, int n_entities, Intersection &is) {
    bool is_hit = false;
    for (int i = 0; i < n_entities; i++) {
        is_hit = entities[i].get_closest_intersection(ray, is) || is_hit;
    }

    // if hit entity has smooth_shading enabled, adjust the normal
    Triangle *tr = is.triangle;
    Entity *e = is.entity;
    if (is_hit && tr != nullptr && e->material.smooth_shading) {
        float u = is.u;
        float v = is.v;
        float w = 1.0f - (u + v);
        vec3 v0_normal = e->d_vertices[tr->idx_a].normal;
        vec3 v1_normal = e->d_vertices[tr->idx_b].normal;
        vec3 v2_normal = e->d_vertices[tr->idx_c].normal;
        is.normal = u * v1_normal + v * v2_normal + w * v0_normal; // pure guess
        is.normal.normalize();
    }

    return is_hit;
}

__device__
bool Entity::get_closest_intersection(const Ray &ray, Intersection &bestHit) {
    switch (this->shape) {
        case SPHERE:
            return get_closest_sphere_intersection(ray, bestHit);
        case TRIANGLE_MESH:
            return get_closest_triangle_mesh_intersection(ray, bestHit);
        default:
            return false;
    }
}

__device__
bool Entity::get_closest_sphere_intersection(const Ray &ray, Intersection &bestHit) {
    vec3 d = ray.origin - this->center;
    float p1 = -dot(ray.direction, d);
    float p2sqr = p1 * p1 - dot(d,d) + this->radius * this->radius;
    if (p2sqr < 0)
        return false;
    float p2 = sqrtf(p2sqr); // sqrt(p2sqr)
    float t = p1 - p2 > 0 ? p1 - p2 : p1 + p2;
    if (t > 0 && t < bestHit.distance)
    {
        bestHit.distance = t;
        bestHit.position = ray.origin + t * ray.direction;
        bestHit.normal = bestHit.position - this->center;
        bestHit.normal.normalize();
        bestHit.entity = this;
        return true;
    }
    return false;
}

__device__
bool Entity::get_closest_triangle_mesh_intersection(const Ray &ray, Intersection &bestHit) {
    if (!this->aabb.intersects(ray))
        return false;
    bool hit = false;
    for (int i = 0; i < this->n_triangles; i++) {
        hit = intersects_triangle(&(this->d_triangles[i]), bestHit, ray) || hit;
    }
    return hit;
}


/*
 * Tomas Akenine-Möller and Ben Trumbore's algorithm.
 *
 * http://fileadmin.cs.lth.se/cs/Personal/Tomas_Akenine-Moller/pubs/raytri_tam.pdf
 */
__device__
bool Entity::intersects_triangle(Triangle *triangle, Intersection &bestHit, const Ray &ray) {
    vec3 v0 = this->d_vertices[triangle->idx_a].position;
    vec3 v1 = this->d_vertices[triangle->idx_b].position;
    vec3 v2 = this->d_vertices[triangle->idx_c].position;
    vec3 e1, e2, pvec, tvec, qvec;
    float t, u, v, det, inv_det;

    e1 = v1 - v0;
    e2 = v2 - v0;

    pvec = cross(ray.direction, e2);
    det = dot(e1, pvec);
    if (fabs(det) < EPSILON) 
        return false;
    
    inv_det = 1.0f / det;
    tvec = ray.origin - v0;
    u = dot(tvec, pvec) * inv_det;
    if (u < 0.0f || u > 1.0f)
        return false;

    qvec = cross(tvec, e1);
    v = dot(ray.direction, qvec) * inv_det;
    if (v < 0.0f || u + v > 1.0f)
        return false;

    t = dot(e2, qvec) * inv_det; 

    if(t > 0 && t < bestHit.distance) {
        bestHit.distance = t;
        bestHit.position = ray.origin + t * ray.direction;
        bestHit.normal = cross(e1, e2).normalized(); // TODO SMOOTH SHADING
        bestHit.entity = this;
        bestHit.triangle = triangle;
        bestHit.u = u;
        bestHit.v = v;
        return true;
    }

    return false;
}

__device__ 
bool AABB::intersects(const Ray &ray) {
    return intersects_aabb(
            this->min.x,
            this->min.y,
            this->min.z,
            this->max.x,
            this->max.y,
            this->max.z,
            ray
    );
}

__device__
bool intersects_aabb(
        float min_x,
        float min_y,
        float min_z,
        float max_x,
        float max_y,
        float max_z,
        const Ray &ray
) {

    float tx1 = (min_x - ray.origin.x)*(1.0f / ray.direction.x);
    float tx2 = (max_x - ray.origin.x)*(1.0f / ray.direction.x);
    float ty1 = (min_y - ray.origin.y)*(1.0f / ray.direction.y);
    float ty2 = (max_y - ray.origin.y)*(1.0f / ray.direction.y);
    float tz1 = (min_z - ray.origin.z)*(1.0f / ray.direction.z);
    float tz2 = (max_z - ray.origin.z)*(1.0f / ray.direction.z);

    float tmin = fminf(tx1, tx2);
    float tmax = fmaxf(tx1, tx2);
    tmin = fmaxf(tmin, fminf(ty1, ty2));
    tmax = fminf(tmax, fmaxf(ty1, ty2));
    tmin = fmaxf(tmin, fminf(tz1, tz2));
    tmax = fminf(tmax, fmaxf(tz1, tz2));
 
    return tmin < tmax;
}