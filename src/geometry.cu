#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include "OBJ_Loader.h"
#include <math.h>
#include <stdlib.h>
#include <iostream>
#include "util.cuh"

//#pragma hd_warning_disable
//#include "glm/glm/glm.hpp"

/************************************************************************************/
/*                                    Constructors                                  */
/************************************************************************************/

Entity::Entity(const std::string &path, const Material &material) {
    objl::Loader ojb_loader;
    bool err_nil = ojb_loader.LoadFile(path);

    // Check for errors or unwelcome extra meshes
    if (!err_nil) {
        std::cerr << "Error loading " << path << std::endl;
        exit(EXIT_FAILURE);
    }
    if (ojb_loader.LoadedMeshes.size() != 1) {
        std::cerr << path << " Does not consist of only 1 mesh" << std::endl;
        exit(EXIT_FAILURE);
    }

    // otherwise get the mesh
    objl::Mesh mesh = ojb_loader.LoadedMeshes[0];

    // allocate space
    this->n_vertices  = mesh.Vertices.size();
    this->n_triangles = mesh.Indices.size() / 3;
    this->vertices  = new Vertex[this->n_vertices];
    this->triangles = new Triangle[this->n_triangles];
    
    // For AABB
    vec3 min(FLT_MAX, FLT_MAX, FLT_MAX);
    vec3 max(-FLT_MAX, -FLT_MAX, -FLT_MAX);

    // get vertices
    for (int i = 0; i < mesh.Vertices.size(); i++) {
        vec3 position(
                mesh.Vertices[i].Position.X,
                mesh.Vertices[i].Position.Y,
                mesh.Vertices[i].Position.Z
        );
        vec3 normal(
                mesh.Vertices[i].Normal.X,
                mesh.Vertices[i].Normal.Y,
                mesh.Vertices[i].Normal.Z
        );
        normal.normalize();

        // update min, max
        min.x = (position.x < min.x) ? position.x : min.x;
        min.y = (position.y < min.y) ? position.y : min.y;
        min.z = (position.z < min.z) ? position.z : min.z;
        max.x = (position.x > max.x) ? position.x : max.x;
        max.y = (position.y > max.y) ? position.y : max.y;
        max.z = (position.z > max.z) ? position.z : max.z;

        this->vertices[i] = Vertex(position, normal);
    }

    // get triangles
    for (int i = 0; i < mesh.Indices.size(); i += 3) {
        this->triangles[i / 3] = Triangle(
                mesh.Indices[i],
                mesh.Indices[i + 1],
                mesh.Indices[i + 2]
        );
    }

    std::cout << "KALAS" << std::endl;
    std::cout << this->vertices[2727].normal << std::endl;
    std::cout << this->vertices[2728].normal << std::endl;
    std::cout << this->vertices[2729].normal << std::endl;

    // debug print all triangle vertices
    /*for(int i = 0; i < this->n_triangles; i++) {
        std::cout << "tri: " << i << std::endl;
        std::cout << this->vertices[this->triangles[i].idx_a].position << std::endl;
        std::cout << this->vertices[this->triangles[i].idx_b].position << std::endl;
        std::cout << this->vertices[this->triangles[i].idx_c].position << std::endl;
    }*/

    // create AABB
    this->aabb = AABB(min, max);

    // set center
    this->center = vec3(
            (min.x + max.x) / 2,
            (min.y + max.y) / 2,
            (min.z + max.z) / 2
    );

    // set shape
    this->shape = TRIANGLE_MESH;
}

Entity::Entity(const vec3 &center, float radius, const Material &material) {
    this->shape     = SPHERE;
    this->center    = center;
    this->radius    = radius;
    this->material  = material;
}

/************************************************************************************/
/*                                 Misc. fucntions                                  */
/************************************************************************************/
void AABB::recalculate(Vertex *vertices, int n_vertices) {
    vec3 min(FLT_MAX, FLT_MAX, FLT_MAX);
    vec3 max(-FLT_MAX, -FLT_MAX, -FLT_MAX);
    for (int i = 0; i < n_vertices; i++) {
        vec3 position = vertices[i].position;
        min.x = (position.x < min.x) ? position.x : min.x;
        min.y = (position.y < min.y) ? position.y : min.y;
        min.z = (position.z < min.z) ? position.z : min.z;
        max.x = (position.x > max.x) ? position.x : max.x;
        max.y = (position.y > max.y) ? position.y : max.y;
        max.z = (position.z > max.z) ? position.z : max.z;
    }
    
    this->min = min;
    this->max = max;
}

void Entity::print() {
    std::cout << "center: " << this->center << std::endl;
    std::cout << "x: " << this->aabb.min.x << " to " << this->aabb.max.x << std::endl;
    std::cout << "y: " << this->aabb.min.y << " to " << this->aabb.max.y << std::endl;
    std::cout << "z: " << this->aabb.min.z << " to " << this->aabb.max.z << std::endl;
}

/************************************************************************************/
/*                            Geometric transformations                             */
/************************************************************************************/

void Entity::scale(float factor) {
    if (this->shape == SPHERE) {
        radius *= factor;
        return;
    }

    if (this->shape == TRIANGLE_MESH) {
        int v_size = this->n_vertices;
        for(int i = 0; i < v_size; i++) {
            vec3 pos = vertices[i].position;
            pos = ((pos - this->center) * factor) + this->center;
            vertices[i].position = pos;
        }

        // recalculate aabb
        this->aabb.recalculate(this->vertices, this->n_vertices);
    }
}

void Entity::translate(vec3 delta) {
    // move center for all shapes
    this->center = this->center + delta;

    if (this->shape == TRIANGLE_MESH) {
        int v_size = this->n_vertices;
        for(int i = 0; i < v_size; i++) {
            vec3 pos = vertices[i].position;
            pos = pos + delta;
            vertices[i].position = pos;
        }

        // recalculate aabb
        this->aabb.recalculate(this->vertices, this->n_vertices);
    }
}

void Entity::rotate(vec3 rot) {
    if (this->shape == SPHERE)
        return;
    
    // rotate on x
    for (int i = 0; i < this->n_vertices; i++) {
        vec3 v = this->vertices[i].position - this->center;
        v = vec3(
            v.x,
            v.y*cos(rot.x) - v.z*sin(rot.x),
            v.y*sin(rot.x) - v.z*cos(rot.x)
        );
        this->vertices[i].position = v + this->center;
    }

    // recalculate aabb
    this->aabb.recalculate(this->vertices, this->n_vertices);

    // TODO rotate on y and z. Preferable not one at a time.
}

/************************************************************************************/
/*                                Memory management                                 */
/************************************************************************************/

void Entity::copy_to_device() {
    if (this->shape == SPHERE)
        return;

    if (this->shape == TRIANGLE_MESH) {
        // copy vertices
        long vertices_size = this->n_vertices * sizeof(Vertex);
        gpuErrchk(hipMalloc(&this->d_vertices, vertices_size));
        hipMemcpy(this->d_vertices, this->vertices, vertices_size, hipMemcpyHostToDevice);
        gpuErrchk(hipPeekAtLastError());

        // copy triangles
        long triangles_size = this->n_triangles * sizeof(Triangle);
        gpuErrchk(hipMalloc(&this->d_triangles, triangles_size));
        hipMemcpy(this->d_triangles, this->triangles, triangles_size, hipMemcpyHostToDevice);
        gpuErrchk(hipPeekAtLastError());
    }
}

void Entity::free_from_device() {
    if (this->shape == SPHERE)
        return;

    // TODO implement mesh case
}


/************************************************************************************/
/*                            Intersection functions                                */
/************************************************************************************/

__device__
bool get_closest_intersection_in_scene(const Ray &ray, Entity *entities, int n_entities, Intersection &is) {
    bool is_hit = false;
    for (int i = 0; i < n_entities; i++) {
        is_hit = entities[i].get_closest_intersection(ray, is) || is_hit;
    }

    // if hit entity has smooth_shading enabled, adjust the normal
    /*Triangle *tr = is.triangle;
    Entity *e = is.entity;
    if (is_hit && tr != nullptr && e->material.smooth_shading) {
        float u = is.u;
        float v = is.v;
        float w = 1.0f - (u + v);
        printf("idxs: %d %d %d\n", tr->idx_a, tr->idx_b, tr->idx_c);
        vec3 v0_normal = e->d_vertices[tr->idx_a].normal;
        vec3 v1_normal = e->d_vertices[tr->idx_b].normal;
        vec3 v2_normal = e->d_vertices[tr->idx_c].normal;
        printf("v0 normal: (%g, %g, %g)\n", v0_normal.x, v0_normal.y, v0_normal.z);
        printf("v1 normal: (%g, %g, %g)\n", v1_normal.x, v1_normal.y, v1_normal.z);
        printf("v2 normal: (%g, %g, %g)\n", v2_normal.x, v2_normal.y, v2_normal.z);
        printf("flat normal: (%g, %g, %g)\n", is.normal.x, is.normal.y, is.normal.z);
        //is.normal = -(u * v1_normal + v * v2_normal + w * v0_normal); // pure guess
        //is.normal = -(u * v2_normal + v * v1_normal + w * v0_normal); // pure guess
        //is.normal = u * v0_normal + v * v2_normal + w * v1_normal; // pure guess
        //is.normal = u * v2_normal + v * v0_normal + w * v1_normal; // pure guess
        //is.normal = u * v1_normal + v * v0_normal + w * v2_normal; // pure guess
        //is.normal = u * v0_normal + v * v1_normal + w * v2_normal; // pure guess
        //is.normal.normalize();
    }*/

    return is_hit;
}

__device__
bool Entity::get_closest_intersection(const Ray &ray, Intersection &bestHit) {
    switch (this->shape) {
        case SPHERE:
            return get_closest_sphere_intersection(ray, bestHit);
        case TRIANGLE_MESH:
            return get_closest_triangle_mesh_intersection(ray, bestHit);
        default:
            return false;
    }
}

__device__
bool Entity::get_closest_sphere_intersection(const Ray &ray, Intersection &bestHit) {
    vec3 d = ray.origin - this->center;
    float p1 = -dot(ray.direction, d);
    float p2sqr = p1 * p1 - dot(d,d) + this->radius * this->radius;
    if (p2sqr < 0)
        return false;
    float p2 = sqrtf(p2sqr); // sqrt(p2sqr)
    float t = p1 - p2 > 0 ? p1 - p2 : p1 + p2;
    if (t > 0 && t < bestHit.distance)
    {
        bestHit.distance = t;
        bestHit.position = ray.origin + t * ray.direction;
        bestHit.normal = bestHit.position - this->center;
        bestHit.normal.normalize();
        bestHit.entity = this;
        return true;
    }
    return false;
}

__device__
bool Entity::get_closest_triangle_mesh_intersection(const Ray &ray, Intersection &bestHit) {
    if (!this->aabb.intersects(ray))
        return false;
    bool hit = false;
    for (int i = 0; i < this->n_triangles; i++) {
        hit = intersects_triangle(&(this->d_triangles[i]), bestHit, ray) || hit;
    }
    return hit;
}


/*
 * Tomas Akenine-Möller and Ben Trumbore's algorithm.
 *
 * http://fileadmin.cs.lth.se/cs/Personal/Tomas_Akenine-Moller/pubs/raytri_tam.pdf
 */
__device__
bool Entity::intersects_triangle(Triangle *triangle, Intersection &bestHit, const Ray &ray) {
    vec3 v0 = this->d_vertices[triangle->idx_a].position;
    vec3 v1 = this->d_vertices[triangle->idx_b].position;
    vec3 v2 = this->d_vertices[triangle->idx_c].position;
    vec3 e1, e2, pvec, tvec, qvec;
    float t, u, v, det, inv_det;

    e1 = v1 - v0;
    e2 = v2 - v0;

    pvec = cross(ray.direction, e2);
    det = dot(e1, pvec);
    if (fabs(det) < EPSILON) 
        return false;
    
    inv_det = 1.0f / det;
    tvec = ray.origin - v0;
    u = dot(tvec, pvec) * inv_det;
    if (u < 0.0f || u > 1.0f)
        return false;

    qvec = cross(tvec, e1);
    v = dot(ray.direction, qvec) * inv_det;
    if (v < 0.0f || u + v > 1.0f)
        return false;

    t = dot(e2, qvec) * inv_det; 

    if(t > 0 && t < bestHit.distance) {
        bestHit.distance = t;
        bestHit.position = ray.origin + t * ray.direction;
        bestHit.normal = cross(e1, e2).normalized(); // TODO SMOOTH SHADING
        bestHit.entity = this;
        bestHit.triangle = triangle;
        bestHit.u = u;
        bestHit.v = v;
        return true;
    }

    return false;
}

__device__ 
bool AABB::intersects(const Ray &ray) {
    return intersects_aabb(
            this->min.x,
            this->min.y,
            this->min.z,
            this->max.x,
            this->max.y,
            this->max.z,
            ray
    );
}

__device__
bool intersects_aabb(
        float min_x,
        float min_y,
        float min_z,
        float max_x,
        float max_y,
        float max_z,
        const Ray &ray
) {

    float tx1 = (min_x - ray.origin.x)*(1.0f / ray.direction.x);
    float tx2 = (max_x - ray.origin.x)*(1.0f / ray.direction.x);
    float ty1 = (min_y - ray.origin.y)*(1.0f / ray.direction.y);
    float ty2 = (max_y - ray.origin.y)*(1.0f / ray.direction.y);
    float tz1 = (min_z - ray.origin.z)*(1.0f / ray.direction.z);
    float tz2 = (max_z - ray.origin.z)*(1.0f / ray.direction.z);

    float tmin = fminf(tx1, tx2);
    float tmax = fmaxf(tx1, tx2);
    tmin = fmaxf(tmin, fminf(ty1, ty2));
    tmax = fminf(tmax, fmaxf(ty1, ty2));
    tmin = fmaxf(tmin, fminf(tz1, tz2));
    tmax = fminf(tmax, fmaxf(tz1, tz2));
 
    return tmin < tmax;
}