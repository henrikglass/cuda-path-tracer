#include "scene.cuh"
#include "util.cuh"

Scene::Scene() {
    this->d_entities = nullptr;
}

void Scene::add_entity(Entity entity) {
    this->entities.push_back(entity);
}

void Scene::copy_to_device() {
    for (int i = 0; i < this->entities.size(); i++) {
        entities[i].copy_to_device();
    }
    long size = this->entities.size()*sizeof(Entity);
    gpuErrchk(hipMalloc(&this->d_entities, size));
    hipMemcpy(this->d_entities, &(this->entities[0]), size, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
}

void Scene::free_from_device() {

}