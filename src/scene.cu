#include "scene.cuh"
#include "util.cuh"

Scene::Scene() {
    this->d_entities = nullptr;
}

void Scene::add_entity(Entity entity) {
    this->entities.push_back(entity);
}

void Scene::copy_to_device() {
    for (Entity e : this->entities) {
        e.copy_to_device();
    }
    long size = this->entities.size()*sizeof(Entity);
    gpuErrchk(hipMalloc(&this->d_entities, size));
    hipMemcpy(this->d_entities, &(this->entities[0]), size, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
}

void Scene::free_from_device() {

}